// CUDA Error abfragen und Dokumentenname und Dokumentenzeile ausgeben
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  cout << " Hit ENTER to exit application." << endl;
	  cin.get();
      if (abort) exit(code);
   }
}
